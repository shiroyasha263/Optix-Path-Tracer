#include "hip/hip_runtime.h"
#include <optix.h>
#include <sutil/vec_math.h>
#include <cuda/helpers.h>
#include "Params.h"

/*! launch parameters in constant memory, filled in by optix upon
    optixLaunch (this gets filled in from the buffer we pass to
    optixLaunch) */
extern "C" __constant__ Params params;

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

static __forceinline__ __device__ void* unpackPointer(unsigned int u0, unsigned int u1) {
    const unsigned long long uptr = static_cast<unsigned long long>(u0) << 32 | u1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void* packPointer(void* ptr, unsigned int& u0, unsigned int& u1) {
    const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
    u0 = uptr >> 32;
    u1 = uptr & 0x00000000ffffffff;
}

//template<typename T>
static __forceinline__ __device__ float3* getPRD() {
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return (reinterpret_cast<float3*>(unpackPointer(u0, u1)));
}

extern "C" __global__ void __closesthit__radiance() {
    printf("This has actually been hit, closest hit\n");
    float3& result = *getPRD();
    result = make_float3(1.0f);
}

extern "C" __global__ void __anyhit__radiance() {
    printf("This has actually been hit, closest hit\n");
    float3& result = *getPRD();
    result = make_float3(1.0f);
}


//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance() {
    float3& result = *getPRD();
    result = make_float3(0.0f);
}



//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const float3      U = params.U;
    const float3      V = params.V;
    const float3      W = params.W;
    const float2      d = 2.0f * make_float2(
        static_cast<float>(idx.x) / static_cast<float>(dim.x),
        static_cast<float>(idx.y) / static_cast<float>(dim.y)
    ) - 1.0f;

    const float3 origin = params.eye;
    const float3 direction = normalize(d.x * U + d.y * V + W);
    float3       payload_rgb = make_float3(0.5f, 0.5f, 0.5f);
    
    unsigned int u0, u1;
    packPointer(&payload_rgb, u0, u1);

    optixTrace(params.handle,
        origin,
        direction,
        0.00f,  // tmin
        1e16f,  // tmax
        0.0f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        0,
        1,
        0,
        u0, u1);

    const uint32_t fbIndex = idx.x + idx.y * params.img_width;
    params.frame_buffer[fbIndex] = make_color(payload_rgb);
}