#include "hip/hip_runtime.h"
#include <optix.h>

#include "Params.h"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>

extern "C" {
    __constant__ Params params;
}



//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

struct RadiancePRD
{
    // TODO: move some state directly into payload registers?
    float3 color;
};


struct Onb
{
    __forceinline__ __device__ Onb(const float3& normal)
    {
        m_normal = normal;

        if (fabs(m_normal.x) > fabs(m_normal.z))
        {
            m_binormal.x = -m_normal.y;
            m_binormal.y = m_normal.x;
            m_binormal.z = 0;
        }
        else
        {
            m_binormal.x = 0;
            m_binormal.y = -m_normal.z;
            m_binormal.z = m_normal.y;
        }

        m_binormal = normalize(m_binormal);
        m_tangent = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3& p) const
    {
        p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};


//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

static __forceinline__ __device__ void* unpackPointer(unsigned int i0, unsigned int i1)
{
    const unsigned long long uptr = static_cast<unsigned long long>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}


static __forceinline__ __device__ void  packPointer(void* ptr, unsigned int& i0, unsigned int& i1)
{
    const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}


static __forceinline__ __device__ RadiancePRD* getPRD()
{
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<RadiancePRD*>(unpackPointer(u0, u1));
}


static __forceinline__ __device__ void setPayloadOcclusion(bool occluded)
{
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}


static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
    // Uniformly sample disk.
    const float r = sqrtf(u1);
    const float phi = 2.0f * M_PIf * u2;
    p.x = r * cosf(phi);
    p.y = r * sinf(phi);

    // Project up to hemisphere.
    p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
}


static __forceinline__ __device__ void traceRadiance(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax,
    RadiancePRD* prd
)
{
    // TODO: deduce stride from num ray-types passed in params

    unsigned int u0, u1;
    packPointer(prd, u0, u1);
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,        // SBT offset
        RAY_TYPE_COUNT,           // SBT stride
        RAY_TYPE_RADIANCE,        // missSBTIndex
        u0, u1);
}


//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
    const int    w = params.width;
    const int    h = params.height;
    const float3 eye = params.eye;
    const float3 U = params.U;
    const float3 V = params.V;
    const float3 W = params.W;
    const uint3  idx = optixGetLaunchIndex();
    const int    subframe_index = params.subframe_index;

    float3 result = make_float3(0.0f);
    int i = params.samples_per_launch;

    const float2 d = 2.0f * make_float2(
        static_cast<float>(idx.x) / static_cast<float>(w),
        static_cast<float>(idx.y) / static_cast<float>(h)
    ) - 1.0f;

    float3 ray_direction = normalize(d.x * U + d.y * V + W);
    float3 ray_origin = eye;

    RadiancePRD prd;
    prd.color = make_float3(0.0f);

    unsigned int u0, u1;
    packPointer(&prd, u0, u1);

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        0.f,
        1e20f,
        0.0f,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        RAY_TYPE_RADIANCE,
        RAY_TYPE_COUNT,
        RAY_TYPE_RADIANCE,
        u0, u1
    );

    const unsigned int image_index = idx.y * params.width + idx.x;
    params.frame_buffer[image_index] = make_color(prd.color);
}


extern "C" __global__ void __miss__radiance()
{
    RadiancePRD* prd = getPRD();
    prd->color = make_float3(0.0f);
}

extern "C" __global__ void __closesthit__radiance()
{
    RadiancePRD* prd = getPRD();
    prd->color = make_float3(1.0f);
}
